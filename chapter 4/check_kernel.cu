#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

__global__ void add(const double *x, const double *y, double *z,const int N);
void check(const double *z, const int N);

int main()
{
    const int N = 100000000;
    
    double *host_x = (double*)malloc(sizeof(double)*N);
    double *host_y = (double*)malloc(sizeof(double)*N);
    double *host_z = (double*)malloc(sizeof(double)*N);

    for(int i = 0; i < N; i++)
    {
        host_x[i] = a;
        host_y[i] = b;
    }
    printf("host_x[0] = %lf, host_y[0] = %lf\n",host_x[0],host_y[0]);

    double *device_x, *device_y, *device_z;
    hipMalloc((void **)&device_x, (sizeof(double)*N));
    hipMalloc((void **)&device_y, (sizeof(double)*N));
    hipMalloc((void **)&device_z, (sizeof(double)*N));

    hipMemcpy(device_x,host_x,sizeof(double)*N,hipMemcpyHostToDevice);
    hipMemcpy(device_y,host_y,sizeof(double)*N,hipMemcpyHostToDevice);

    // 错误的设置线程块数 线程块数最大是1024 
    const int block_size = 1280;
    int grid_size;
    if(N % block_size == 0) grid_size = N/block_size;
    else grid_size = N/block_size+1;

    printf("block_size = %d, grid_size = %d, thread_nums = %d\n",
                            block_size,grid_size,block_size*grid_size);

    add<<<grid_size,block_size>>>(device_x,device_y,device_z, N);
    
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    hipMemcpy(host_z,device_z,sizeof(double)*N,hipMemcpyDeviceToHost);
    printf("host_z[0] = %lf\n",host_z[0]);
    check(host_z,N);

    free(host_x);
    free(host_y);
    free(host_z);
    
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_z);

    return 0;
}

__global__ void add(const double *x, const double *y, double *z, const int N)
{
    // i = 128*blockIdx.x + threadIdx.x
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i <  N)    
    {
        z[i] = x[i] + y[i];
    }
    else return;
}
void check(const double *z, const int N)
{
    bool check_flag = false;
    for(int i = 0; i < N; i++)
    {
        if(fabs(z[i]-c) > EPSILON)
        {
            check_flag = true;
            printf("check z[%d] = %lf\n",i,z[i]);
            printf("%s\n", check_flag ? "Has errors" : "No errors");
            return ;
        }
    }
    printf("%s\n", check_flag ? "Has errors" : "No errors");
}